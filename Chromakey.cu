#include "hip/hip_runtime.h"
#include <iostream>
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>

using namespace std;
using namespace cv;

#define CHECK(value) {                                          \
    hipError_t _m_cudaStat = value;                                        \
    if (_m_cudaStat != hipSuccess) {                                       \
        cout<< "Error:" << hipGetErrorString(_m_cudaStat) \
            << " at line " << __LINE__ << " in file " << __FILE__ << "\n"; \
        exit(1);                                                            \
    } }

#define TILE_WIDTH 8

__global__ void setBackColor(uchar3 * pxs, uchar3* back, int length, int minHue, int maxHue)
{
    int i = threadIdx.x + blockIdx.x*blockDim.x;

    if (i < length)
    {
        uchar *p = (uchar*)(&pxs[i]);
        uchar *bp = (uchar*)(&back[i]);

        double r = p[2] / 255.0, g = p[1] / 255.0, b = p[0] / 255.0;
        double cmax = r > g ? r : g;
        cmax = cmax > b ? cmax : b;

        double cmin = r < g ? r : g;
        cmin = cmin < b ? cmin : b;

        double delta = cmax - cmin;

        double h;

        if (delta < 0.0001 && delta > -0.0001)
          h = 0.0;
        else if (cmax == r)
        {
            h = 60 * ((int)((g - b) / delta) % 6);
        }
        else if (cmax == g)
        {
            h = 60 * (((b - r) / delta) + 2);
        }
        else
        {
            h = 60 * (((r - g) / delta) + 4);
        }

        if (h >= minHue && h < maxHue)
        {
            p[0] = bp[0];
            p[1] = bp[1];
            p[2] = bp[2];
        }
    }
}

int main(int argc, char* argv[])
{
    VideoCapture cap(argv[0]);
    VideoCapture backCap(argv[1]);

    if(!cap.isOpened() || !backCap.isOpened()) return 0;

    Mat frame, back;
    VideoWriter outputVideo;

    int minHue, maxHue;
    sscanf(argv[3], "%d", &minHue);
    sscanf(argv[4], "%d", &maxHue);

    int width = 640;
    int height = 360;
    int fps = 24;

    namedWindow("1");
    namedWindow("2");

    Size winSize(width, height);
    outputVideo.open(argv[2], CV_FOURCC('M','J','P','G'), fps, winSize, true);

    if(!outputVideo.isOpened()) return 0;

    uchar3 *devFrame, *devBack;

    CHECK(  hipMalloc(&devFrame, 3 * width * height)  );
    CHECK(  hipMalloc(&devBack, 3 * width * height)  );

    while(true)
    {
        if (!cap.read(frame)) break;

        int N = frame.rows * frame.cols;
        uchar3 *frameData = (uchar3*)frame.data;
        uchar3 *backData;
        if(backCap.read(back))
          backData = (uchar3*)back.data;
        else
        {
            backData = frameData;
            cout << "back";
        }

        CHECK( hipMemcpy(devFrame, frameData, 3* N,hipMemcpyHostToDevice) );
        CHECK( hipMemcpy(devBack, backData, 3* N,hipMemcpyHostToDevice) );

        imshow("1", frame);

        setBackColor<<<(N + TILE_WIDTH - 1) / TILE_WIDTH, TILE_WIDTH>>>(devFrame, devBack, N, minHue, maxHue);
        CHECK(  hipMemcpy(frameData, devFrame, 3*N,hipMemcpyDeviceToHost)  );
        CHECK(  hipGetLastError()  );
        imshow("2", frame);

        outputVideo.write(frame);
        waitKey(20);
    }

    hipFree(&devFrame);
    hipFree(&devBack);

    return 0;
}
